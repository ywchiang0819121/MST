#include "hip/hip_runtime.h"
#include "globals.h"

//void init() { 
//	for (int i = 0; i<V; ++i) p[i] = i; 
//}

int find(int x) { 
	return x == p[x] ? x : (p[x] = find(p[x])); 
}

void uni(int x, int y) {
	p[find(x)] = find(y); 
}

__device__ int finden(int x) {
	return x == cudaptrp[x] ? x : (cudaptrp[x] = finden(cudaptrp[x]));
}

__device__ void dieunio(int x,int y) {
	cudaptrp[finden(x)] = finden(y);
}

//struct suche
//{
//	__host__ __device__
//	int operator()(int &_x) {
//		return _x == cudap[_x] ? _x : (cudap[_x] = find(cudap[_x]));
//	}
//};
//
//struct dieuni
//{
//	__host__ __device__
//	void operator()(int x, int y) {
//		cudap[find(x)] = find(y);
//	}
//};
//
//struct funktion
//{
//	__host__ __device__
//	void operator()(bounder &_x) {
//		int a = suchen(_x.a);
//		int b = suchen(_x.b);
//		int c = _x.c;
//		if (a == b) return;
//		cross_edge += 1;
//		if (c < minweight[a] || c == minweight[a] && i < selectedbounders[a]) {
//			minweight[a] = c;
//			selectedbounders[a] = i;
//		}
//
//		if (c < minweight[b] || c == minweight[b] && i < selectedbounders[b]) {
//			minweight[b] = c;
//			selectedbounders[b] = i;
//		}
//	}
//};

__global__ void funktion(bounder *cedge) {
	int i = blockIdx.x*blockDim.x + blockIdx.x;
	int a = cedge[i].a;
	int b = cedge[i].b;
	int c = cedge[i].c;
	if (c < cudaptrmw[a] || c == cudaptrmw[a] && i < cudaptrsb[a]) {
		cudaptrmw[a] = c;
		cudaptrsb[a] = i;
	}
	if (c < cudaptrmw[a] || c == cudaptrmw[a] && i < cudaptrsb[a]) {
		cudaptrmw[a] = c;
		cudaptrsb[a] = i;
	}
}

void Boruvka()
{
	thrust::copy(first, first + 4, cudap.begin());
	bounder* cudaptredge;
	cudaptredge = thrust::raw_pointer_cast(cudaedge.data());
	cudaptrsb = thrust::raw_pointer_cast(cudaselectedbounders.data());
	cudaptrp = thrust::raw_pointer_cast(cudap.data());
	cudaptrmw = thrust::raw_pointer_cast(cudaminweight.data());
	//funktion <<<5, 1 >>> (bounder cudaptredge);
	while (true)
	{
		cross_edge = 0;
		thrust::fill(cudaminweight.begin(), cudaminweight.end(), 1e9);
		funktion << <5, 1 >> > (bounder cudaptredge);
		if (cross_edge == 0) break;
		for (int i = 0; i<V; ++i)
			if (minweight[i] != 1e9)
				uni(edge[selectedbounders[i]].a, edge[selectedbounders[i]].b);
		break;
	}
	vector<int> tmp(V);
	thrust::copy(selectedbounders.begin(), selectedbounders.end(), tmp.begin());
	set<int> finalroute(tmp.begin(), tmp.end());
	int allweight = 0;
	cout << "selected bounders:" << endl;
	for (auto i : finalroute) {
		cout << "vertex(a)=" << edge[i].a << "\tvertex(b)=" << edge[i].b << "\tbouder weight(c)=" << edge[i].c << endl;
		allweight += edge[i].c;
	}
	cout << "total weight:" << allweight << endl;
}

int main() {
	cudaedge.push_back({ 0,1,10 });
	cudaedge.push_back({ 0,2,6 });
	cudaedge.push_back({ 0,3,5 });
	cudaedge.push_back({ 2,3,4 });
	cudaedge.push_back({ 1,3,15 });
	edge.push_back({ 0,1,10 });
	edge.push_back({ 0,2,6 });
	edge.push_back({ 0,3,5 });
	edge.push_back({ 2,3,4 });
	edge.push_back({ 1,3,15 });
	Boruvka();
	system("pause");
	return 0;
}